#include <iostream>
#include <engine/engine.h>
#include <engine/cuda_func.h>
#include <arrow/api.h>
#include <arrow/csv/api.h>
#include <arrow/io/file.h>
#include <arrow/result.h>
#include <iostream>


#define N 3

int main() {
    float *a, *b, *out; 
    float *d_a, *d_b, *d_out;

    a   = (float*)malloc(sizeof(float) * N);
    b   = (float*)malloc(sizeof(float) * N);
    out = (float*)malloc(sizeof(float) * N);

    for(int i = 0; i < N; i++){
        a[i] = 1.0f; 
        b[i] = 2.0f;
        printf("iter: a: %f, b: %f\n", a[i], b[i]);
    }

    hipMalloc((void**)&d_a, sizeof(float) * N);
    hipMalloc((void**)&d_b, sizeof(float) * N);
    hipMalloc((void**)&d_out, sizeof(float) * N);

    hipMemcpy(d_a, a, sizeof(float) * N, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, sizeof(float) * N, hipMemcpyHostToDevice);

    matAddition<<<1, 1>>>(d_out, d_a, d_b, N);

    hipMemcpy(out, d_out, sizeof(float) * N, hipMemcpyDeviceToHost);

    for (int i = 0; i < N; i++){
        printf("%f ", out[i]);
    }
    printf("\n");

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_out);

    free(a);
    free(b);
    free(out);

    hipDeviceSynchronize();

    void *ptr;
    int i = 5;
    std::cout << "i: " << i << std::endl;
    std::cout << "&i: " << &i << std::endl;
    std::cout << "ptr: " << ptr << std::endl;
    std::cout << "&ptr: " << &ptr << std::endl;

    ptr = &i;
    std::cout << "ptr: " << ptr << std::endl;
    std::cout << "&ptr: " << &ptr << std::endl;

    return 0;
}
