#include "hip/hip_runtime.h"
#include <engine.h>

__global__ void cuda_hello(){
    printf("Hello World from GPU!\n");
}

__global__ void matAddition(float *out, float *a, float *b, int n) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < n) {
        out[index] = a[index] + b[index];
    }
}